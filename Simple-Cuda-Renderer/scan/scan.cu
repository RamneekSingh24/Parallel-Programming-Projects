#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256

#define NUM_CORES 512 // GTX 1050 Ti (mobile) has 760 cuda cores

// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void up_sweep_kernel(int *result, int two_d, int two_dplus1)
{
    int myId = blockIdx.x * blockDim.x + threadIdx.x;
    int myIdx = myId * two_dplus1;
    result[myIdx + two_dplus1 - 1] += result[myIdx + two_d - 1];
};

__global__ void down_sweep_kernel(int *result, int two_d, int two_dplus1)
{
    int myId = blockIdx.x * blockDim.x + threadIdx.x;
    int myIdx = myId * two_dplus1;
    int t = result[myIdx + two_d - 1];
    result[myIdx + two_d - 1] = result[myIdx + two_dplus1 - 1];
    result[myIdx + two_dplus1 - 1] += t;
};

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
// In place exclusive_scan in result
void exclusive_scan([[unused]] int *input, int N, int *result)
{

    const int rounded_length = nextPow2(N);

    //  upsweep phase
    for (int two_d = 1; two_d <= rounded_length / 2; two_d *= 2)
    {
        int two_dplus1 = 2 * two_d;

        int num_threads = rounded_length / two_dplus1;
        int threads_per_block = THREADS_PER_BLOCK;
        int num_blocks = num_threads / THREADS_PER_BLOCK;

        if (num_threads < THREADS_PER_BLOCK)
        {
            num_blocks = 1;
            threads_per_block = num_threads;
        }

        up_sweep_kernel<<<num_blocks, threads_per_block>>>(result, two_d, two_dplus1);
        // calls b/w consecutive kernals are automatically syncronized
    }

    hipDeviceSynchronize();

    hipMemset(result + (rounded_length - 1), 0, sizeof(int));

    // down sweep phase
    for (int two_d = rounded_length / 2; two_d >= 1; two_d /= 2)
    {
        int two_dplus1 = 2 * two_d;

        int num_threads = rounded_length / two_dplus1;
        int threads_per_block = THREADS_PER_BLOCK;
        int num_blocks = num_threads / THREADS_PER_BLOCK;

        if (num_threads < THREADS_PER_BLOCK)
        {
            num_blocks = 1;
            threads_per_block = num_threads;
        }

        down_sweep_kernel<<<num_blocks, threads_per_block>>>(result, two_d, two_dplus1);
        // calls b/w consecutive kernals are automatically syncronized
    }
}

//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int *inarray, int *end, int *resultarray)
{
    int *device_result;
    int *device_input;
    int N = end - inarray;

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);

    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both tdevice_inputhe input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;

    hipFree(device_input);
    hipFree(device_result);

    return overallDuration;
}

// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int *inarray, int *end, int *resultarray)
{

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration;
}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found

__global__ void set_kernel(int *device_input, int *device_output, int len)
{
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (myIdx >= len - 1)
        return;
    if (device_input[myIdx] == device_input[myIdx + 1])
    {
        device_output[myIdx] = 1;
    }
    else
    {
        device_output[myIdx] = 0;
    }
}

__global__ void make_result_kernel(int *device_input, int *device_output, int len)
{
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (myIdx >= len - 1)
        return;
    int resIdx = device_input[myIdx];
    if (device_input[myIdx + 1] - resIdx == 1)
    {
        device_output[resIdx] = myIdx;
    }
}

int find_repeats(int *device_input, int length, int *device_output)
{

    int rounded_length = nextPow2(length);
    int num_blocks = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    set_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(device_input, device_output, length);

    exclusive_scan(device_input, length, device_output);

    hipDeviceSynchronize();
    int count;
    hipMemcpy(&count, device_output + length - 1, sizeof(int), hipMemcpyDeviceToHost);

    make_result_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(device_output, device_input, length);

    hipMemcpy(device_output, device_input, count * sizeof(int), hipMemcpyDeviceToDevice);

    return count;
}

//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length)
{

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);

    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();

    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime;
    return duration;
}

void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("shared mem: %d\n", deviceProps.sharedMemPerBlock / 1024);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
